
#include <hip/hip_runtime.h>
#ifdef __NVCC__

/**
 * BUDE CUDA kernel file
 **/

#define CUDIFY_BOOST_CONTEXT_STACK_SIZE 32768

//#define SE_CLASS1

#include <float.h>
#include <stdio.h>
#include <sys/time.h>

#include "Vector/map_vector.hpp"
#include "util/stat/common_statistics.hpp"

//#define USE_SHARED

constexpr int pos = 0;
constexpr int ind = 1;

constexpr int x = 0;
constexpr int y = 1;
constexpr int z = 2;

constexpr int   hbtype = 0;
constexpr int radius = 1;
constexpr int hphb = 2;
constexpr int elsc = 3;

#ifndef NUM_TD_PER_THREAD
// Good for CPU
//#define NUM_TD_PER_THREAD 256
// Good for GPU
#define NUM_TD_PER_THREAD 4
#endif

typedef struct
{
    float x, y, z;
    int   index;
} Atom;

typedef struct
{
    int   hbtype;
    float radius;
    float hphb;
    float elsc;
} FFParams;

typedef struct
{
    int    natlig;
    int    natpro;
    int    ntypes;
    int    nposes;
    char     * deckDir;
    int iterations;
} Params;

Params params;

typedef struct
{
  // _lin = AOS
  openfpm::vector_gpu_lin<aggregate<float[3],int>> d_protein;
  // AOS
  openfpm::vector_gpu_lin<aggregate<float[3],int>> d_ligand;
  // AOS
  openfpm::vector_gpu_lin<aggregate<int,float,float,float>> d_forcefield;
  // SOA
  openfpm::vector_gpu<aggregate<float>> d_results;
  // SOA
  openfpm::vector_gpu<aggregate<float,float,float,float,float,float>> d_poses;
  openfpm::vector<double> gflops_data;

    int deviceIndex;
    int wgsize;
    int posesPerWI;
} OpenFPM;


double getTimestamp()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_usec + tv.tv_sec*1e6;
}

void printTimings(double start, double end, double poses_per_wi, openfpm::vector<double> & gflops_data)
{
    double ms = ((end-start)/params.iterations)*1e-3;

    // Compute FLOP/s
    double runtime   = ms*1e-3;
    double ops_per_wi = 27*poses_per_wi
        + params.natlig*(3 + 18*poses_per_wi + params.natpro*(11 + 30*poses_per_wi))
        + poses_per_wi;
    double total_ops     = ops_per_wi * (params.nposes/poses_per_wi);
    double flops      = total_ops / runtime;
    double gflops     = flops / 1e9;

    gflops_data.add(gflops);

    double interactions         =
        (double)params.nposes
        * (double)params.natlig
        * (double)params.natpro;
    double interactions_per_sec = interactions / runtime;

    // Print stats
    printf("- Total time:     %7.2lf ms\n", (end-start)*1e-3);
    printf("- Average time:   %7.2lf ms\n", ms);
    printf("- Interactions/s: %7.2lf billion\n", (interactions_per_sec / 1e9));
    printf("- GFLOP/s:        %7.2lf\n", gflops);
}

// Numeric constants
#define ZERO    0.0f
#define QUARTER 0.25f
#define HALF    0.5f
#define ONE     1.0f
#define TWO     2.0f
#define FOUR    4.0f
#define CNSTNT 45.0f

#define HBTYPE_F 70
#define HBTYPE_E 69

// The data structure for one atom - 16 bytes

typedef struct
{
  float x, y, z, w;
} Transform;

#define HARDNESS 38.0f
#define NPNPDIST  5.5f
#define NPPDIST   1.0f

__device__ inline void compute_transformation_matrix(const float transform_0,
    const float transform_1,
    const float transform_2,
    const float transform_3,
    const float transform_4,
    const float transform_5,
    Transform* transform)
{
  const float sx = sin(transform_0);
  const float cx = cos(transform_0);
  const float sy = sin(transform_1);
  const float cy = cos(transform_1);
  const float sz = sin(transform_2);
  const float cz = cos(transform_2);

  transform[0].x = cy*cz;
  transform[0].y = sx*sy*cz - cx*sz;
  transform[0].z = cx*sy*cz + sx*sz;
  transform[0].w = transform_3;
  transform[1].x = cy*sz;
  transform[1].y = sx*sy*sz + cx*cz;
  transform[1].z = cx*sy*sz - sx*cz;
  transform[1].w = transform_4;
  transform[2].x = -sy;
  transform[2].y = sx*cy;
  transform[2].z = cx*cy;
  transform[2].w = transform_5;
}



template<typename vector_atom, typename vector_ff, typename vector_tr, typename vector_out>
__global__ void fasten_main(const int natlig,
    const int natpro,
    const vector_atom protein_molecule,
    const vector_atom ligand_molecule,
    const vector_tr transforms,
    vector_out etotals,
    const vector_ff global_forcefield,
    const int num_atom_types,
    const int numTransforms)
{
  // Get index of first TD
  int ix = blockIdx.x*blockDim.x*NUM_TD_PER_THREAD + threadIdx.x;

  // Have extra threads do the last member intead of return.
  // A return would disable use of barriers, so not using return is better
  ix = ix < numTransforms ? ix : numTransforms - NUM_TD_PER_THREAD;

#ifdef USE_SHARED
  __shared__ FFParams forcefield[100];
  if(ix < num_atom_types)
  {
    forcefield[ix].hbtype = global_forcefield.template get<hbtype>(ix);
    forcefield[ix].radius = global_forcefield.template get<radius>(ix);
    forcefield[ix].hphb = global_forcefield.template get<hphb>(ix);
    forcefield[ix].elsc = global_forcefield.template get<elsc>(ix);

  }
#else
#endif

  // Compute transformation matrix to private memory
  float etot[NUM_TD_PER_THREAD];
  Transform transform[NUM_TD_PER_THREAD][3];
  const int lsz = blockDim.x;
  #pragma omp simd
  for (int i = 0; i < NUM_TD_PER_THREAD; i++)
  {
    int index = ix + i*lsz;
    compute_transformation_matrix(
        transforms.template get<0>(index),
        transforms.template get<1>(index),
        transforms.template get<2>(index),
        transforms.template get<3>(index),
        transforms.template get<4>(index),
        transforms.template get<5>(index),
        transform[i]);
    etot[i] = ZERO;
  }

#ifdef USE_SHARED
  __syncthreads();
#endif

  // Loop over ligand atoms
  int il = 0;
  do
  {
    // Load ligand atom data
    const Atom l_atom = {ligand_molecule.template get<pos>(il)[x],
    			 ligand_molecule.template get<pos>(il)[y],
    			 ligand_molecule.template get<pos>(il)[z],
    			 ligand_molecule.template get<ind>(il)}; 

    const FFParams l_params = {global_forcefield.template get<hbtype>(l_atom.index),
    			       global_forcefield.template get<radius>(l_atom.index),
    			       global_forcefield.template get<hphb>(l_atom.index),
    			       global_forcefield.template get<elsc>(l_atom.index)};
    const bool lhphb_ltz = l_params.hphb<ZERO;
    const bool lhphb_gtz = l_params.hphb>ZERO;

    float lpos_x[NUM_TD_PER_THREAD];
    float lpos_y[NUM_TD_PER_THREAD];
    float lpos_z[NUM_TD_PER_THREAD];
    const float4 linitpos = make_float4(l_atom.x,l_atom.y,l_atom.z,ONE);
    #pragma omp simd
    for (int i = 0; i < NUM_TD_PER_THREAD; i++)
    {
      // Transform ligand atom
      lpos_x[i] = transform[i][0].w + linitpos.x*transform[i][0].x + 
        linitpos.y*transform[i][0].y + linitpos.z*transform[i][0].z;
      lpos_y[i] = transform[i][1].w + linitpos.x*transform[i][1].x + 
        linitpos.y*transform[i][1].y + linitpos.z*transform[i][1].z;
      lpos_z[i] = transform[i][2].w + linitpos.x*transform[i][2].x + 
        linitpos.y*transform[i][2].y + linitpos.z*transform[i][2].z;
    }

    // Loop over protein atoms
    int ip = 0;
    do
    {
      // Load protein atom data
      const Atom p_atom = {protein_molecule.template get<pos>(ip)[x],
      			   protein_molecule.template get<pos>(ip)[y],
      			   protein_molecule.template get<pos>(ip)[z],
      			   protein_molecule.template get<ind>(ip)};

      const FFParams p_params = {global_forcefield.template get<hbtype>(p_atom.index),
	      			 global_forcefield.template get<radius>(p_atom.index),
				 global_forcefield.template get<hphb>(p_atom.index),
				 global_forcefield.template get<elsc>(p_atom.index)};

      const float radij   = p_params.radius + l_params.radius;
      const float r_radij = 1.0f/radij;

      const float elcdst  = (p_params.hbtype==HBTYPE_F && l_params.hbtype==HBTYPE_F) ? FOUR    : TWO;
      const float elcdst1 = (p_params.hbtype==HBTYPE_F && l_params.hbtype==HBTYPE_F) ? QUARTER : HALF;
      const bool type_E   = ((p_params.hbtype==HBTYPE_E || l_params.hbtype==HBTYPE_E));

      const bool phphb_ltz = p_params.hphb<ZERO;
      const bool phphb_gtz = p_params.hphb>ZERO;
      const bool phphb_nz  = p_params.hphb!=ZERO;
      const float p_hphb   = p_params.hphb * (phphb_ltz && lhphb_gtz ? -ONE : ONE);
      const float l_hphb   = l_params.hphb * (phphb_gtz && lhphb_ltz ? -ONE : ONE);
      const float distdslv = (phphb_ltz ? (lhphb_ltz ? NPNPDIST : NPPDIST) : (lhphb_ltz ? NPPDIST : -FLT_MAX) );

      float r_distdslv = 1.0f/distdslv;

      const float chrg_init = l_params.elsc * p_params.elsc;
      const float dslv_init = p_hphb + l_hphb;

      #pragma omp simd
      for (int i = 0; i < NUM_TD_PER_THREAD; i++)
      {
        // Calculate distance between atoms
        const float x      = lpos_x[i] - p_atom.x;
        const float y      = lpos_y[i] - p_atom.y;
        const float z      = lpos_z[i] - p_atom.z;
        const float distij = sqrtf(x*x + y*y + z*z);

        // Calculate the sum of the sphere radii
        const float distbb = distij - radij;
        const bool  zone1  = (distbb < ZERO);

        // Calculate steric energy
        etot[i] += (ONE - (distij*r_radij)) * (zone1 ? 2*HARDNESS : ZERO);

        // Calculate formal and dipole charge interactions
        float chrg_e = chrg_init * ((zone1 ? 1 : (ONE - distbb*elcdst1)) 
            * (distbb<elcdst ? 1 : ZERO));
        const float neg_chrg_e = -fabs(chrg_e);
        chrg_e = type_E ? neg_chrg_e : chrg_e;
        etot[i] += chrg_e*CNSTNT;

        // Calculate the two cases for Nonpolar-Polar repulsive interactions
        const float coeff  = (ONE - (distbb *r_distdslv));
        float dslv_e = dslv_init * ((distbb<distdslv && phphb_nz) ? 1 : ZERO);
        dslv_e *= (zone1 ? 1 : coeff);
        etot[i] += dslv_e;
      }
    } 
    while (++ip < natpro); // loop over protein atoms
  } 
  while (++il < natlig); // loop over ligand atoms

  // Write results
  const int td_base = blockIdx.x*blockDim.x*NUM_TD_PER_THREAD + threadIdx.x;
  if (td_base < numTransforms)
  {
    #pragma omp simd
    for (int i = 0; i < NUM_TD_PER_THREAD; i++)
    {
      etotals.template get<0>(td_base+i*blockDim.x) = etot[i]*HALF;
    }
  }
} //end of fasten_main


void runCUDA(OpenFPM & _openfpm)
{
  _openfpm.d_protein.hostToDevice<pos,ind>();
  _openfpm.d_ligand.hostToDevice<pos,ind>();
  _openfpm.d_forcefield.hostToDevice<hbtype,radius,hphb,elsc>();
  _openfpm.d_results.resize(params.nposes);
  _openfpm.d_poses.template hostToDevice<0,1,2,3,4,5>();

  size_t global = ceil(params.nposes/(double)_openfpm.posesPerWI);
  global = ceil(global/(double)_openfpm.wgsize);
  size_t local  = _openfpm.wgsize;
  size_t shared = params.ntypes * sizeof(FFParams);

  cudaDeviceSynchronize();

  double start = getTimestamp();

  for(int ii = 0; ii < params.iterations; ++ii)
  {

    CUDA_LAUNCH_DIM3(fasten_main,global, local,
        params.natlig, 
        params.natpro,
        _openfpm.d_protein.toKernel(),
        _openfpm.d_ligand.toKernel(),
        _openfpm.d_poses.toKernel(),
        _openfpm.d_results.toKernel(),
        _openfpm.d_forcefield.toKernel(),
        params.ntypes,
        params.nposes);

  }

  cudaDeviceSynchronize();

  double end = getTimestamp();

  _openfpm.d_results.deviceToHost<0>();

  printTimings(start, end, _openfpm.posesPerWI, _openfpm.gflops_data);
}

#define MAX_PLATFORMS     8
#define MAX_DEVICES      32
#define MAX_INFO_STRING 256

#define DATA_DIR          "bm1"
#define FILE_LIGAND       "/ligand.in"
#define FILE_PROTEIN      "/protein.in"
#define FILE_FORCEFIELD   "/forcefield.in"
#define FILE_POSES        "/poses.in"
#define FILE_REF_ENERGIES "/ref_energies.out"

#define REF_NPOSES 65536

// Energy evaluation parameters
#define CNSTNT   45.0f
#define HBTYPE_F 70
#define HBTYPE_E 69
#define HARDNESS 38.0f
#define NPNPDIST  5.5f
#define NPPDIST   1.0f

void printTimings(double start, double end, double poses_per_wi);
void checkError(int err, const char *op);

FILE* openFile(const char *parent, const char *child,
               const char* mode, long *length)
{
  char name[strlen(parent) + strlen(child) + 1];
  strcpy(name, parent);
  strcat(name, child);

  FILE *file = NULL;
  if (!(file = fopen(name, mode)))
  {
    fprintf(stderr, "Failed to open '%s'\n", name);
    exit(1);
  }
  if(length){
    fseek(file, 0, SEEK_END);
    *length = ftell(file);
    rewind(file);
  }
  return file;
}

int parseInt(const char *str)
{
  char *next;
  int value = strtoul(str, &next, 10);
  return strlen(next) ? -1 : value;
}

void loadParameters(int argc, char *argv[], OpenFPM & _openfpm)
{
  // Defaults
  params.deckDir        = DATA_DIR;
  params.iterations = 8;
  _openfpm.wgsize      = 256;
  _openfpm.posesPerWI  = NUM_TD_PER_THREAD;
  int nposes        = 65536;

  for (int i = 1; i < argc; i++)
  {
    if (!strcmp(argv[i], "--device") || !strcmp(argv[i], "-d"))
    {
      if (++i >= argc || (_openfpm.deviceIndex = parseInt(argv[i])) < 0)
      {
        printf("Invalid device index\n");
        exit(1);
      }
    }
    else if (!strcmp(argv[i], "--iterations") || !strcmp(argv[i], "-i"))
    {
      if (++i >= argc || (params.iterations = parseInt(argv[i])) < 0)
      {
        printf("Invalid number of iterations\n");
        exit(1);
      }
    }
    else if (!strcmp(argv[i], "--numposes") || !strcmp(argv[i], "-n"))
    {
      if (++i >= argc || (nposes = parseInt(argv[i])) < 0)
      {
        printf("Invalid number of poses\n");
        exit(1);
      }
    }
    else if (!strcmp(argv[i], "--posesperwi") || !strcmp(argv[i], "-p"))
    {
      if (++i >= argc || (_openfpm.posesPerWI = parseInt(argv[i])) < 0)
      {
        printf("Invalid poses-per-workitem value\n");
        exit(1);
      }
    }
    else if (!strcmp(argv[i], "--wgsize") || !strcmp(argv[i], "-w"))
    {
      if (++i >= argc || (_openfpm.wgsize = parseInt(argv[i])) < 0)
      {
        printf("Invalid work-group size\n");
        exit(1);
      }
    }
    else if (!strcmp(argv[i], "--deck"))
    {
      if (++i >= argc)
      {
        printf("Invalid deck\n");
        exit(1);
      }
      params.deckDir = argv[i];
    }
    else if (!strcmp(argv[i], "--help") || !strcmp(argv[i], "-h"))
    {
      printf("\n");
      printf("Usage: ./bude [OPTIONS]\n\n");
      printf("Options:\n");
      printf("  -h  --help               Print this message\n");
      printf("      --list               List available devices\n");
      printf("      --device     INDEX   Select device at INDEX\n");
      printf("  -i  --iterations I       Repeat kernel I times\n");
      printf("  -n  --numposes   N       Compute results for N poses\n");
      printf("  -p  --poserperwi PPWI    Compute PPWI poses per work-item\n");
      printf("  -w  --wgsize     WGSIZE  Run with work-group size WGSIZE\n");
      printf("      --deck       DECK    Use the DECK directory as input deck\n");
      printf("\n");
      exit(0);
    }
    else
    {
      printf("Unrecognized argument '%s' (try '--help')\n", argv[i]);
      exit(1);
    }
  }

  FILE *file = NULL;
  long length;

  file = openFile(params.deckDir, FILE_LIGAND, "rb", &length);
  params.natlig = length / sizeof(Atom);
  _openfpm.d_ligand.resize(params.natlig);

  for (int i = 0 ; i < _openfpm.d_ligand.size() ; i++)
  {
	  fread(&_openfpm.d_ligand.template get<pos>(i)[0],sizeof(float),1,file);
	  fread(&_openfpm.d_ligand.template get<pos>(i)[1],sizeof(float),1,file);
	  fread(&_openfpm.d_ligand.template get<pos>(i)[2],sizeof(float),1,file);
	  fread(&_openfpm.d_ligand.template get<ind>(i),sizeof(int),1,file);
  }

  fclose(file);

  file = openFile(params.deckDir, FILE_PROTEIN, "rb", &length);
  params.natpro = length / sizeof(Atom);

  _openfpm.d_protein.resize(params.natpro);

  for (int i = 0 ; i < _openfpm.d_protein.size() ; i++)
  {
          fread(&_openfpm.d_protein.template get<pos>(i)[0],sizeof(float),1,file);
          fread(&_openfpm.d_protein.template get<pos>(i)[1],sizeof(float),1,file);
          fread(&_openfpm.d_protein.template get<pos>(i)[2],sizeof(float),1,file);
          fread(&_openfpm.d_protein.template get<ind>(i),sizeof(int),1,file);
  }

  fclose(file);

  file = openFile(params.deckDir, FILE_FORCEFIELD, "rb", &length);
  params.ntypes = length / sizeof(FFParams);

  _openfpm.d_forcefield.resize(params.ntypes);

  for (int i = 0 ; i < _openfpm.d_forcefield.size() ; i++)
  {
          fread(&_openfpm.d_forcefield.template get<hbtype>(i),sizeof(int),1,file);
          fread(&_openfpm.d_forcefield.template get<radius>(i),sizeof(float),1,file);
          fread(&_openfpm.d_forcefield.template get<hphb>(i),sizeof(float),1,file);
          fread(&_openfpm.d_forcefield.template get<elsc>(i),sizeof(float),1,file);
  }

  fclose(file);

  file = openFile(params.deckDir, FILE_POSES, "rb", &length);
  _openfpm.d_poses.resize(nposes);

  long available = length / 6 / sizeof(float);
  params.nposes = 0;
  while (params.nposes < nposes)
  {
    long fetch = nposes - params.nposes;
    if (fetch > available)
      fetch = available;

      fseek(file, 0*available*sizeof(float), SEEK_SET);
      for (int k = 0 ; k < fetch ; k++)
      {fread(&_openfpm.d_poses.template get<0>(params.nposes+k),sizeof(float),1,file);}

      fseek(file, 1*available*sizeof(float), SEEK_SET);
      for (int k = 0 ; k < fetch ; k++)
      {fread(&_openfpm.d_poses.template get<1>(params.nposes+k),sizeof(float),1,file);}

      fseek(file, 2*available*sizeof(float), SEEK_SET);
      for (int k = 0 ; k < fetch ; k++)
      {fread(&_openfpm.d_poses.template get<2>(params.nposes+k),sizeof(float),1,file);}

      fseek(file, 3*available*sizeof(float), SEEK_SET);
      for (int k = 0 ; k < fetch ; k++)
      {fread(&_openfpm.d_poses.template get<3>(params.nposes+k),sizeof(float),1,file);}

      fseek(file, 4*available*sizeof(float), SEEK_SET);
      for (int k = 0 ; k < fetch ; k++)
      {fread(&_openfpm.d_poses.template get<4>(params.nposes+k),sizeof(float),1,file);}

      fseek(file, 5*available*sizeof(float), SEEK_SET);
      for (int k = 0 ; k < fetch ; k++)
      {fread(&_openfpm.d_poses.template get<5>(params.nposes+k),sizeof(float),1,file);}


    rewind(file);

    params.nposes += fetch;
  }
  fclose(file);
}

#if !defined(__APPLE__) && !defined(__powerpc64__)
#include <fenv.h>
#include <xmmintrin.h>
#include <pmmintrin.h>
#endif

int main(int argc, char *argv[])
{
#if !defined(__APPLE__) && !defined(__powerpc64__)
  _MM_SET_FLUSH_ZERO_MODE(_MM_FLUSH_ZERO_ON);
  _MM_SET_DENORMALS_ZERO_MODE(_MM_DENORMALS_ZERO_ON);
#endif
  init_wrappers();

  OpenFPM _openfpm;
  loadParameters(argc, argv, _openfpm);
  printf("\n");
  printf("Poses     : %d\n", params.nposes);
  printf("Iterations: %d\n", params.iterations);
  printf("Ligands   : %d\n", params.natlig);
  printf("Proteins  : %d\n", params.natpro);
  printf("Deck      : %s\n", params.deckDir);
  float *resultsRef = (float *)malloc(params.nposes*sizeof(float));

  // We run the benchmark 30 times to get mean and variace
  for (int i = 0 ; i < 30 ; i++)
  {
    printf("Iteration %d\n",i);

    runCUDA(_openfpm);
  }

  // calculate mean and variance
  double mean;
  double dev;
  standard_deviation(_openfpm.gflops_data,mean,dev);

  printf("\n\n\nMean %f ~ %f GFlops/s \n\n\n",mean,dev);
  FILE* perf_out = openFile("./","performance_out", "w", NULL);
  char out[256];
  sprintf(out,"%f %f",mean,dev);
  fwrite(out,1,strlen(out),perf_out);
  fclose(perf_out);

  // Load reference results from file
  FILE* ref_energies = openFile(params.deckDir, FILE_REF_ENERGIES, "r", NULL);
  size_t n_ref_poses = params.nposes;
  if (params.nposes > REF_NPOSES) {
    printf("Only validating the first %d poses.\n", REF_NPOSES);
    n_ref_poses = REF_NPOSES;
  }

  for (size_t i = 0; i < n_ref_poses; i++)
    fscanf(ref_energies, "%f", &resultsRef[i]);

  fclose(ref_energies);

  float maxdiff = -100.0f;
  printf("\n Reference        CUDA   (diff)\n");
  for (int i = 0; i < n_ref_poses; i++)
  {
    if (fabs(resultsRef[i]) < 1.f && fabs(_openfpm.d_results.template get<0>(i)) < 1.f) continue;

    float diff = fabs(resultsRef[i] - _openfpm.d_results.template get<0>(i)) / _openfpm.d_results.template get<0>(i);
    if (diff > maxdiff) {
      maxdiff = diff;
      // printf ("Maxdiff: %.2f (%.3f vs %.3f)\n", maxdiff, resultsRef[i], resultsCUDA[i]);
    }

    if (i < 8)
      printf("%7.2f    vs   %7.2f  (%5.2f%%)\n", resultsRef[i], _openfpm.d_results.template get<0>(i), 100*diff);
  }
  printf("\nLargest difference was %.3f%%\n\n", maxdiff*100);

  free(resultsRef);
}

#else

int main(int argc, char *argv[])
{
}

#endif

