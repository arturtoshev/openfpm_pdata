
#include <hip/hip_runtime.h>
#ifdef __NVCC__

#include "Vector/map_vector.hpp"
#include "util/stat/common_statistics.hpp"

#define NELEMENTS 67108864

//! Memory bandwidth with small calculations
template<typename vector_type, typename vector_type2>
__global__ void translate_fill_prop_write(vector_type vd_out, vector_type2 vd_in)
{
	auto p = blockIdx.x * blockDim.x + threadIdx.x;

	float a = vd_in.template get<0>(p)[0];

	vd_out.template get<0>(p) = a;

	vd_out.template get<1>(p)[0] = a;
	vd_out.template get<1>(p)[1] = a;

	vd_out.template get<2>(p)[0][0] = a;
	vd_out.template get<2>(p)[0][1] = a;
	vd_out.template get<2>(p)[1][0] = a;
    vd_out.template get<2>(p)[1][1] = a;
    vd_in.template get<0>(p)[1] = a;
}

template<typename vector_type, typename vector_type2>
__global__ void translate_fill_prop_read(vector_type vd_out, vector_type2 vd_in)
{
	auto p = blockIdx.x * blockDim.x + threadIdx.x;

	float a = vd_out.template get<0>(p);

	float b = vd_out.template get<1>(p)[0];
	float c = vd_out.template get<1>(p)[1];

	float d = vd_out.template get<2>(p)[0][0];
	float e = vd_out.template get<2>(p)[0][1];
	float f = vd_out.template get<2>(p)[1][0];
	float g = vd_out.template get<2>(p)[1][1];
    
	float h = vd_in.template get<0>(p)[0];
	vd_in.template get<0>(p)[1] = a+b+c+d+e+f+g+h;
}


template<typename in_type, typename out_type>
void check_write(in_type & in, out_type & out)
{
    out.template deviceToHost<0,1,2>();
    in.template deviceToHost<0>();

    bool success = true;
    for (int i = 0 ; i < NELEMENTS; i++)
    {
        float a = in.template get<0>(i)[0];

        success &= out.template get<0>(i) == a;

        success &= out.template get<1>(i)[0] == a;
        success &= out.template get<1>(i)[1] == a;

        success &= out.template get<2>(i)[0][0] == a;
        success &= out.template get<2>(i)[0][1] == a;
        success &= out.template get<2>(i)[1][0] == a;
        success &= out.template get<2>(i)[1][1] == a;

        success &= in.template get<0>(i)[1] == a;
    }

    if (success == false)
    {
            std::cout << "FAIL WRITE" << std::endl;
            exit(1);
    }
}

template<typename in_type, typename out_type>
void check_read(in_type & in, out_type & out)
{
    out.template deviceToHost<0,1,2>();
    in.template deviceToHost<0>();

    bool success = true;
    for (int i = 0 ; i < NELEMENTS ; i++)
    {
        float a = out.template get<0>(i);

        float b = out.template get<1>(i)[0];
        float c = out.template get<1>(i)[1];

        float d = out.template get<2>(i)[0][0];
        float e = out.template get<2>(i)[0][1];
        float f = out.template get<2>(i)[1][0];
        float g = out.template get<2>(i)[1][1];

        float h = in.template get<0>(i)[0];

        success &= in.template get<0>(i)[1] == (a+b+c+d+e+f+g+h);

        if (success == false)
        {
            std::cout << "FAIL READ " << i << "   " << in.template get<0>(i)[1] << " != " << a+b+c+d+e+f+g+h << std::endl;
            exit(1);
        }
    }
}

template<typename vector_type, typename vector_type2>
__global__ void initialize_buff(vector_type vd_out, vector_type2 vd_in)
{
	auto i = blockIdx.x * blockDim.x + threadIdx.x;

    vd_in.template get<0>(i)[0] = i;
    vd_in.template get<0>(i)[1] = i+100.0;

    vd_out.template get<0>(i) = i+200.0;

    vd_out.template get<1>(i)[0] = i;
    vd_out.template get<1>(i)[1] = i+100.0;

    vd_out.template get<2>(i)[0][0] = i;
    vd_out.template get<2>(i)[0][1] = i+100.0;
    vd_out.template get<2>(i)[1][0] = i+200.0;
    vd_out.template get<2>(i)[1][1] = i+300.0;
}

template<typename vin_type, typename vout_type>
void initialize_buf(vin_type in, vout_type out)
{
    auto ite = out.getGPUIterator(256);
    CUDA_LAUNCH(initialize_buff,ite,out.toKernel(),in.toKernel());
}

int main(int argc, char *argv[])
{
    init_wrappers();

    openfpm::vector_gpu<aggregate<float,float[2],float[2][2]>> out;
    openfpm::vector_gpu<aggregate<float[2]>> in;

    int nele = NELEMENTS;

    out.resize(nele);
    in.resize(nele);

    initialize_buf(in,out);

    // Read write test with TLS

    auto ite = out.getGPUIterator(256);

    openfpm::vector<double> res;
    res.resize(100);

    for (int i = 0 ; i < 110 ; i++)
    {
        cudaDeviceSynchronize();
        timer t;
        t.start();


        CUDA_LAUNCH(translate_fill_prop_write,ite,out.toKernel(),in.toKernel());

        cudaDeviceSynchronize();

        t.stop();

        if (i >=10)
        {res.get(i-10) = (double)nele*4*9 / t.getwct() * 1e-9;}

        std::cout << "Time: " << t.getwct() << std::endl;
        std::cout << "BW: " << (double)nele*4*9 / t.getwct() * 1e-9 << " GB/s"  << std::endl;
    }

    double mean_write_tls = 0.0;
    double dev_write_tls = 0.0;
    standard_deviation(res,mean_write_tls,dev_write_tls);

    check_write(in,out);

    initialize_buf(in,out);

    for (int i = 0 ; i < 110 ; i++)
    {
        cudaDeviceSynchronize();
        timer t;
        t.start();


        CUDA_LAUNCH(translate_fill_prop_read,ite,out.toKernel(),in.toKernel());

        cudaDeviceSynchronize();

        t.stop();

        if (i >=10)
        {res.get(i-10) = (double)nele*4*9 / t.getwct() * 1e-9;}

        std::cout << "Time: " << t.getwct() << std::endl;
        std::cout << "BW: " << (double)nele*4*9 / t.getwct() * 1e-9 << " GB/s"  << std::endl;
    }

    double mean_read_tls = 0.0;
    double dev_read_tls = 0.0;
    standard_deviation(res,mean_read_tls,dev_read_tls);

    check_read(in,out);

    //////////////

    /////////////////////////////////////////// LAMBDA //////////////////////////////////////////

    initialize_buf(in,out);

    for (int i = 0 ; i < 110 ; i++)
    {
        cudaDeviceSynchronize();
        timer t;
        t.start();

        auto vd_out = out.toKernel();
        auto vd_in = in.toKernel();

        auto lamb = [vd_out,vd_in] __device__ (dim3 & blockIdx, dim3 & threadIdx)
        {
            auto p = blockIdx.x * blockDim.x + threadIdx.x;

            float a = vd_in.template get<0>(p)[0];

            vd_out.template get<0>(p) = a;

            vd_out.template get<1>(p)[0] = a;
            vd_out.template get<1>(p)[1] = a;
        
            vd_out.template get<2>(p)[0][0] = a;
            vd_out.template get<2>(p)[0][1] = a;
            vd_out.template get<2>(p)[1][0] = a;
            vd_out.template get<2>(p)[1][1] = a;
            vd_in.template get<0>(p)[1] = a;
        };

        CUDA_LAUNCH_LAMBDA(ite, lamb);

        cudaDeviceSynchronize();

        t.stop();

        if (i >=10)
        {res.get(i-10) = (double)nele*4*9 / t.getwct() * 1e-9;}

        std::cout << "Time: " << t.getwct() << std::endl;
        std::cout << "BW: " << (double)nele*4*9 / t.getwct() * 1e-9 << " GB/s"  << std::endl;
    }

    double mean_write_lamb = 0.0;
    double dev_write_lamb = 0.0;
    standard_deviation(res,mean_write_lamb,dev_write_lamb);

    initialize_buf(in,out);

    for (int i = 0 ; i < 110 ; i++)
    {
        cudaDeviceSynchronize();
        timer t;
        t.start();


        auto vd_out = out.toKernel();
        auto vd_in = in.toKernel();

        auto lamb = [vd_out,vd_in] __device__ (dim3 & blockIdx, dim3 & threadIdx)
                            {
                                auto p = blockIdx.x * blockDim.x + threadIdx.x;

                                float a = vd_out.template get<0>(p);

                                float b = vd_out.template get<1>(p)[0];
                                float c = vd_out.template get<1>(p)[1];
                            
                                float d = vd_out.template get<2>(p)[0][0];
                                float e = vd_out.template get<2>(p)[0][1];
                                float f = vd_out.template get<2>(p)[1][0];
                                float g = vd_out.template get<2>(p)[1][1];
                                
                                float h = vd_in.template get<0>(p)[0];
                                vd_in.template get<0>(p)[1] = a+b+c+d+e+f+g+h;
                            };

        CUDA_LAUNCH_LAMBDA(ite, lamb);

        cudaDeviceSynchronize();

        t.stop();

        if (i >=10)
        {res.get(i-10) = (double)nele*4*9 / t.getwct() * 1e-9;}

        std::cout << "Time: " << t.getwct() << std::endl;
        std::cout << "BW: " << (double)nele*4*9 / t.getwct() * 1e-9 << " GB/s"  << std::endl;
    }

    double mean_read_lamb = 0.0;
    double dev_read_lamb = 0.0;
    standard_deviation(res,mean_read_lamb,dev_read_lamb);

    // Array benchmark

    for (int i = 0 ; i < 110 ; i++)
    {
        cudaDeviceSynchronize();
        timer t;
        t.start();

	    float * out_s = (float *)out.getDeviceBuffer<0>();
	    float * out_v = (float *)out.getDeviceBuffer<1>();
	    float * out_m = (float *)out.getDeviceBuffer<2>();
        float * in_v = (float *)in.getDeviceBuffer<0>();
        
        int stride = out.capacity();

        auto lamb_arr_write = [out_s,out_v,out_m,in_v,stride] __device__ (dim3 & blockIdx, dim3 & threadIdx)
        {
            auto p = blockIdx.x * blockDim.x + threadIdx.x;

            float a = in_v[p + 0*stride];
        
            out_s[p] = a;
        
            out_v[p + 0*stride] = a;
            out_v[p + 1*stride] = a;
        
            out_m[p + 0*2*stride + 0*stride ] = a;
            out_m[p + 0*2*stride + 1*stride ] = a;
            out_m[p + 1*2*stride + 0*stride ] = a;
            out_m[p + 1*2*stride + 1*stride ] = a;
            in_v[p + 1*stride] = a;
        };

        CUDA_LAUNCH_LAMBDA(ite,lamb_arr_write);

        cudaDeviceSynchronize();

        t.stop();

        if (i >=10)
        {res.get(i-10) = (double)nele*4*9 / t.getwct() * 1e-9;}

        std::cout << "Time ARR: " << t.getwct() << std::endl;
        std::cout << "BW ARR: " << (double)nele*4*9 / t.getwct() * 1e-9 << " GB/s"  << std::endl;
    }

    double mean_write_arr = 0.0;
    double dev_write_arr = 0.0;
    standard_deviation(res,mean_write_arr,dev_write_arr);

    check_write(in,out);

    for (int i = 0 ; i < 110 ; i++)
    {
        cudaDeviceSynchronize();
        timer t;
        t.start();

	    float * out_s = (float *)out.getDeviceBuffer<0>();
	    float * out_v = (float *)out.getDeviceBuffer<1>();
	    float * out_m = (float *)out.getDeviceBuffer<2>();
        float * in_v = (float *)in.getDeviceBuffer<0>();
        
        int stride = out.capacity();

        auto lamb_arr_red = [out_s,out_v,out_m,in_v,stride] __device__ (dim3 & blockIdx, dim3 & threadIdx)
        {
            auto p = blockIdx.x * blockDim.x + threadIdx.x;

            float a = out_s[p];
        
            float b = out_v[p + 0*stride];
            float c = out_v[p + 1*stride];
        
            float d = out_m[p + 0*2*stride + 0*stride];
            float e = out_m[p + 0*2*stride + 1*stride];
            float f = out_m[p + 1*2*stride + 0*stride];
            float g = out_m[p + 1*2*stride + 1*stride];
            
            float h = in_v[p + 0*stride];
            in_v[p + 1*stride] = a+b+c+d+e+f+g+h;
        };

        CUDA_LAUNCH_LAMBDA(ite,lamb_arr_red);

        cudaDeviceSynchronize();

        t.stop();

        if (i >=10)
        {res.get(i-10) = (double)nele*4*9 / t.getwct() * 1e-9;}

        std::cout << "Time ARR: " << t.getwct() << std::endl;
        std::cout << "BW ARR: " << (double)nele*4*9 / t.getwct() * 1e-9 << " GB/s"  << std::endl;
    }

    double mean_read_arr = 0.0;
    double dev_read_arr = 0.0;
    standard_deviation(res,mean_read_arr,dev_read_arr);

    check_read(in,out);

    ///////////////////

    #ifdef CUDIFY_USE_CUDA

    for (int i = 0 ; i < 110 ; i++)
    {
        cudaDeviceSynchronize();
        timer t;
        t.start();

        float * a = (float *)in.getDeviceBuffer<0>();
        float * b = (float *)out.getDeviceBuffer<1>();

        cudaMemcpy(a,b,2*NELEMENTS*4,cudaMemcpyDeviceToDevice);

        cudaDeviceSynchronize();

        t.stop();

        if (i >=10)
        {res.get(i-10) = (double)nele*4*4 / t.getwct() * 1e-9;}

        std::cout << "Time: " << t.getwct() << std::endl;
        std::cout << "BW: " << (double)nele*4*4 / t.getwct() * 1e-9 << " GB/s"  << std::endl;
    }    

    double mean_read_mes = 0.0;
    double dev_read_mes = 0.0;
    standard_deviation(res,mean_read_mes,dev_read_mes);

    std::cout << "Average measured: " << mean_read_mes << "  deviation: " << dev_read_mes << std::endl;

    #endif

    std::cout << "Average READ with TLS: " << mean_read_tls << "  deviation: " << dev_read_tls << std::endl;
    std::cout << "Average WRITE with TLS: " << mean_write_tls << "  deviation: " << dev_write_tls << std::endl;

    std::cout << "Average READ with lamb: " << mean_read_lamb << "  deviation: " << dev_read_lamb << std::endl;
    std::cout << "Average WRITE with lamb: " << mean_write_lamb << "  deviation: " << dev_write_lamb << std::endl;

    std::cout << "Average WRITE with array: " << mean_write_arr << "  deviation: " << dev_write_arr << std::endl;
    std::cout << "Average READ with array: " << mean_read_arr << "  deviation: " << dev_read_arr << std::endl;
}

#else

int main(int argc, char *argv[])
{
}

#endif

