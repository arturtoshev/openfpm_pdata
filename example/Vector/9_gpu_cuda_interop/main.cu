
#include <hip/hip_runtime.h>
/*! \page Vector_9_gpu_cuda_interop Vector 9 GPU cuda interoperability
 *
 *
 * [TOC]
 *
 *
 * # GPU CUDA inter-operability with plain arrays # {#GPU_9_cuda_interop}
 *
 * OpenFPM provide the possibility to operate with CUDA using plain arrays. In particular we can ask to the distributed
 * data-structure to return a CUDA device pointer to the data. Before operate with such pointer we must understand
 * how vector_dist_gpu store data internally in order to correctly read data from such pointer
 *
 * ## Array striding {#e9_array_stride}
 *
 * To understand how vector_dist_gpu store data, we will print the address in memory of each element. Let start for printing
 *  the address of the first particle for all the properties
 *
 * \snippet Vector/9_gpu_cuda_interop/main.cu first_particle_prop_zero_and_one_two
 *
 * Running the program on one process we get
 *
 * \code
First particle property 0, address: 0x7f8d63c00400
First particle property 1, address: 0x7f8d63c00600
First particle property 2, address: 0x7f8d83400000
 * \endcode
 *
 * As we can see the scalar property, vector and tensor properties are not nearly contiguous, the reason is that every properties use
 * its own CUDA buffer, and each property can be off-loaded separately.
 *
 * Now we check how the component of the vector are stored in memory, printing the address of the components for the vector and for
 * the tensor property.
 *
 * \snippet Vector/9_gpu_cuda_interop/main.cu first_particle_vector_tensor_layout
 *
 * The output that we can obtain is something like
 *
  \code
Capacity internal vector: 128
First particle property 1 component 0, address: 0x7f8d63c00600
First particle property 1 component 1, address: 0x7f8d63c00800
First particle property 2 component 00, address: 0x7f8d83400000
First particle property 2 component 01, address: 0x7f8d83400200
First particle property 2 component 10, address: 0x7f8d83400400
First particle property 2 component 11, address: 0x7f8d83400600
Second particle property 1 component 0, address: 0x7f8d63c00604
Second particle property 1 component 1, address: 0x7f8d63c00804
Second particle property 2 component 00, address: 0x7f8d83400004
Second particle property 2 component 01, address: 0x7f8d83400204
Second particle property 2 component 10, address: 0x7f8d83400404
Second particle property 2 component 11, address: 0x7f8d83400604
  \endcode
 *
 * As we can see the vector property of first particle component y is not contiguous to x, but is 0x200 = 4 byte * 128 offset from
 * the component x. What is contiguous to particle 0 component x is particle 1 component x
 *
 * \note This is in general hidden and transparent to the user. Infact in the example we have shown, we were able to create a distributed vector and
 *       compute on it without know how vector_dist store data. It only become necessary if you want to use CUDA with plain primitive arrays
 *
 * There is a reason why vector_dist_gpu use this layout and is because of memory coalesced access. Suppose you want to access
 * a vector property in the GPU kernel like this
 *
 * \code
 *
 * vd.template getProp<vector>(p)[0]
 *
 * \endcode
 *
 * In general what we do is to map the particle index p to a GPU thread that handle that particle. Doing so let see what happen
 * when one SM hit that instruction using the standard layout.
 *
 * \verbatim
                          Memory                                                              Memory

   particle 0  [0]x        0x000      <------- Access thread 0         particle 0  [0]x        0x000      <------- Access thread 0
               [1]y        0x004                                       particle 1  [0]x        0x004      <------- Access thread 1
   particle 1  [0]x        0x008      <------- Access thread 1         particle 2  [0]x        0x008      <------- Access thread 2
               [1]y          .                                         particle 3  [0]x        0x00C      <------- Access thread 3
   particle 2  [0]x          .                                                  .
               [1]y          .                                                  .
               .             .                                                  .
               .             .                                                  .
               .             .                                                  .
   particle N  [0]x          .       <-------- Access thread N                  .
               [1]y          .                                         particle 0  [1]y

                  Case A                                                            Case B
 * \endverbatim
 *
 * As we can see from the image in case A there is a jump of 4 byte compared of Case B. And this mean that the instruction will read double
 *  of the cache lines compared to case B.
 *
 *  Remain to understand why having 100 particles the component y stay at 4 * 128 = 512 byte instead of 4 * 100 = 400 byte. One power 2 alignment
 *  the other is instead related to the internal preallocated vector buffer. Suppose to have a vector with 4 particles and we want to add one
 *  particle at the end. Because we do not have space in theory we have to create a vector of 5 elements, copy the 4 elements in the new vector
 *  and add the last elements. This is clearly expensive when the vector become big, copy the full vector to just one element would not make sense.
 *  OpenFPM use by default a policy to expand the vector by a factor (default = 2) to guarantee that if a vector with N elements starting from
 *  an a vector of size 0 have cost O(N).
 *
 * \note OpenFPM by default does not operate any attempt to expand the virtual address space of the structure to avoid copy
 *
 * ## Interoperability with CUDA {#e9_interop_cuda}
 *
 * Now that we understood the structure of the device pointer, we can see how we can use the internal device pointer in a cuda kernel.
 * We now launch a kernel just to print the information inside the buffer. To get the device CUDA pointer we can use the combo functions
 * \b getPropVector() \b to the the internal propetries vector follow  by \b getDeviceBuffer<0>() \b that return the CUDA device
 * buffer for the property 0
 *
 * \snippet Vector/9_gpu_cuda_interop/main.cu print_50
 *
 * the kernel print the information of particle 50. To note how we pass primitive arrays to the kernel and we use capacity to
 * access the component of vector and the tensor accordingly  to what we explained in array striding
 *
 * \snippet Vector/9_gpu_cuda_interop/main.cu print_data_kernel
 *
 *
 *
 * ## Full code ## {#code_e9_sim}
 *
 * \include Vector/9_gpu_cuda_interop/main.cu
 *
 */

#ifdef __NVCC__

#include "Vector/vector_dist.hpp"

//! [print_data_kernel]

__global__ void print_data_particle_50(float * scalar, float * vector, float * tensor, int capacity)
{
	int p = threadIdx.x + blockIdx.x * blockDim.x;

	if (p == 50)
	{
		printf("Scalar particle %d = %f\n",p,scalar[p]);

		printf("Vector particle %d = %f\n",p,vector[p]);
		printf("Vector particle %d = %f\n",p,vector[p + capacity]);

		printf("Tensor particle %d = %f\n",p,tensor[p + (0*2 + 0)*capacity]);
		printf("Tensor particle %d = %f\n",p,tensor[p + (0*2 + 1)*capacity]);
		printf("Tensor particle %d = %f\n",p,tensor[p + (1*2 + 0)*capacity]);
		printf("Tensor particle %d = %f\n",p,tensor[p + (1*2 + 1)*capacity]);
	}
}

//! [print_data_kernel]

int main(int argc, char* argv[])
{
    // initialize the library
	openfpm_init(&argc,&argv);

	// Here we define our domain a 2D box with internals from 0 to 1.0 for x and y
	Box<2,float> domain({0.0,0.0},{1.0,1.0});

	// Here we define the boundary conditions of our problem
    size_t bc[2]={PERIODIC,PERIODIC};

	// extended boundary around the domain, and the processor domain
	Ghost<2,float> g(0.05);

    vector_dist_gpu<2,float, aggregate<float,float[2],float[2][2]> > vd(100,domain,bc,g);

	auto it = vd.getDomainIterator();

	while (it.isNext())
	{
		auto p = it.get();

		// we define x, assign a random position between 0.0 and 1.0
		vd.getPos(p)[0] = (float)rand() / RAND_MAX;

		// we define y, assign a random position between 0.0 and 1.0
		vd.getPos(p)[1] = (float)rand() / RAND_MAX;

		vd.template getProp<0>(p) = vd.getPos(p)[0] + vd.getPos(p)[1];

		vd.template getProp<1>(p)[0] = vd.getPos(p)[0];
		vd.template getProp<1>(p)[1] = vd.getPos(p)[1];

		vd.template getProp<2>(p)[0][0] = vd.getPos(p)[0];
		vd.template getProp<2>(p)[0][1] = vd.getPos(p)[1];
		vd.template getProp<2>(p)[1][0] = vd.getPos(p)[0] + vd.getPos(p)[1];
		vd.template getProp<2>(p)[1][1] = vd.getPos(p)[1] - vd.getPos(p)[0];

		// next particle
		++it;
	}

	vd.map();

	//! \cond [map_and_ghost_get_on_gpu] \endcond

	//! \cond [first_particle_prop_zero_and_one_two] \endcond

	std::cout << "First particle property 0, address: " << &vd.template getProp<0>(0) << std::endl;
	std::cout << "First particle property 1, address: " << &vd.template getProp<1>(0)[0] << std::endl;
	std::cout << "First particle property 2, address: " << &vd.template getProp<2>(0)[0][0] << std::endl;

	//! \cond [first_particle_prop_zero_and_one_two] \endcond

	//! \cond [first_particle_vector_tensor_layout] \endcond

	std::cout << "Capacity internal vector: " << vd.getPropVector().capacity() << std::endl;

	std::cout << "First particle property 1 component 0, address: " << &vd.template getProp<1>(0)[0] << std::endl;
	std::cout << "First particle property 1 component 1, address: " << &vd.template getProp<1>(0)[1] << std::endl;

	std::cout << "First particle property 2 component 00, address: " << &vd.template getProp<2>(0)[0][0] << std::endl;
	std::cout << "First particle property 2 component 01, address: " << &vd.template getProp<2>(0)[0][1] << std::endl;
	std::cout << "First particle property 2 component 10, address: " << &vd.template getProp<2>(0)[1][0] << std::endl;
	std::cout << "First particle property 2 component 11, address: " << &vd.template getProp<2>(0)[1][1] << std::endl;

	std::cout << "Second particle property 1 component 0, address: " << &vd.template getProp<1>(1)[0] << std::endl;
	std::cout << "Second particle property 1 component 1, address: " << &vd.template getProp<1>(1)[1] << std::endl;

	std::cout << "Second particle property 2 component 00, address: " << &vd.template getProp<2>(1)[0][0] << std::endl;
	std::cout << "Second particle property 2 component 01, address: " << &vd.template getProp<2>(1)[0][1] << std::endl;
	std::cout << "Second particle property 2 component 10, address: " << &vd.template getProp<2>(1)[1][0] << std::endl;
	std::cout << "Second particle property 2 component 11, address: " << &vd.template getProp<2>(1)[1][1] << std::endl;

	//! \cond [first_particle_vector_tensor_layout] \endcond

	std::cout << std::endl;

	//! \cond [print_50] \endcond

	vd.template hostToDeviceProp<0,1,2>();

	CUDA_LAUNCH_DIM3(print_data_particle_50,100,1,(float *)vd.getPropVector().template getDeviceBuffer<0>(),
			               (float *)vd.getPropVector().template getDeviceBuffer<1>(),
			               (float *)vd.getPropVector().template getDeviceBuffer<2>(),
			               vd.getPropVector().capacity());

	//! \cond [print_50] \endcond

	openfpm_finalize();
}

#else

int main(int argc, char* argv[])
{
        return 0;
}

#endif
